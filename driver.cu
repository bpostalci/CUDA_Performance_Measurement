// Berat Postalcioglu
/*OUTPUT

	blocksPerGrid   threadsPerBlock   time to generate
	-------------   ---------------   ----------------
		 157           256              0.04400000 ms.
		  79           512              0.05434880 ms.
		  40          1024              0.09233920 ms.
	   40000             1              0.00174080 ms.
	   20000             1              0.00174080 ms.
	   10000             1              0.00179200 ms.
		5000             1              0.00189440 ms.
		1000             1              0.07427360 ms.
		   2             1             27.85148430 ms.
		   1             1             54.44231415 ms.

*/
#include "hip/hip_runtime.h"

#include <ctime>
#include <cmath>
#include <iostream>
#include <cstdio>

using namespace std;

const int ArrSize = 40000;


__global__ void diffGPU(double *a, double *b, double *c, int size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < size)
	{
		c[tid] = a[tid] - b[tid];
		tid += blockDim.x * gridDim.x;
	}
}


void generateArray(double *arr, double size)
{
	for (int i = 0; i < size; i++)
	{
		arr[i] = rand() % 100 + 1;
	}
}

double sumArray(double *data, int count)
{
	double result = 0;
	for (int i = 0; i < count; i++)
	{
		result += data[i];
	}
	return result;
}

double sumArrayDiff(double *a, double *b, int count)
{
	double *c=new double[count];
	for (int i = 0; i < count; i++)
	{
		c[i] = a[i] - b[i];
	}

	double result = sumArray(c, count);
	delete[] c;

	return result;
}

int* diffCPU(int *v1, int *v2)
{
	int res[ArrSize];
	for (int i = 0; i < ArrSize; i++)
	{
		res[i] = v1[i] - v2[i];
	}
	return res;
}

void displayArray(int *arr, int size)
{
	for (int i = 0; i < size; i++)
	{
		std::cout << arr[i] << " ";
	}
	std::cout << std::endl;
}

float duration(double *devA, double *devB, double *devC, int blocksPerGrid, int threadsPerBlock, double resultCPU)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	// gpu work
	diffGPU <<<threadsPerBlock, blocksPerGrid >>> (devA, devB, devC, ArrSize);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	double *resFromGpu = new double[ArrSize];
	hipMemcpy(resFromGpu, devC, sizeof(double)*ArrSize, hipMemcpyDeviceToHost);
	double sum = sumArray(resFromGpu, ArrSize);

	delete[] resFromGpu;

	if (sum != resultCPU)
	{
		cout << "Results from CPU and GPU are not same... Exiting..." << endl;
		exit(1);
	}

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	return elapsedTime;
}

float averageDuration(double *devA, double *devB, double *devC,
	int blocksPerGrid, int threadsPerBlock, double resultCPU, int repetition)
{
	double totalElapsedTime = 0;
	for (int i = 0; i < repetition; i++)
	{
		totalElapsedTime += duration(devA, devB, devC, blocksPerGrid, threadsPerBlock, resultCPU);
	}
	return (totalElapsedTime / repetition);
}

void prepareTable(double *devA, double *devB, double *devC, int blocksPerGrid, int threadsPerBlock, double resultCPU, int repetition) {

	double ave = averageDuration(devA, devB, devC, blocksPerGrid, threadsPerBlock, resultCPU, repetition);
	printf("%8d", blocksPerGrid);
	printf("%14d", threadsPerBlock);
	printf("%24.8f", ave);
	printf(" ms.\n");
}

int main()
{
	srand(time(NULL));

	// cpu
	double a[ArrSize], b[ArrSize], c[ArrSize];
	generateArray(a, ArrSize);
	generateArray(b, ArrSize);
	double resultCPU = sumArrayDiff(a, b, ArrSize);

	// gpu
	unsigned int totalBytes = ArrSize * sizeof(double);
	double *devA, *devB, *devC;
	hipMalloc(&devA, totalBytes);
	hipMalloc(&devB, totalBytes);
	hipMalloc(&devC, totalBytes);

	hipMemcpy(devA, a, totalBytes, hipMemcpyHostToDevice);
	hipMemcpy(devB, b, totalBytes, hipMemcpyHostToDevice);

	cout << "blocksPerGrid" << "   " << "threadsPerBlock" << "   " << "time to generate" << endl;
	cout << "-------------" << "   " << "---------------" << "   " << "----------------" << endl;
	prepareTable(devA, devB, devC, 157, 256, resultCPU, 1);
	prepareTable(devA, devB, devC, 79, 512, resultCPU, 20);
	prepareTable(devA, devB, devC, 40, 1024, resultCPU, 20);
	prepareTable(devA, devB, devC, 40000, 1, resultCPU, 20);
	prepareTable(devA, devB, devC, 20000, 1, resultCPU, 20);
	prepareTable(devA, devB, devC, 10000, 1, resultCPU, 20);
	prepareTable(devA, devB, devC, 5000, 1, resultCPU, 20);
	prepareTable(devA, devB, devC, 1000, 1, resultCPU, 20);
	prepareTable(devA, devB, devC, 2, 1, resultCPU, 20);
	prepareTable(devA, devB, devC, 1, 1, resultCPU, 20);

	return 0;
}